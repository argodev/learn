
#include <hip/hip_runtime.h>
#define W 500
#define H 500
#define TX 32 // number of threads per block along x-axis
#define TY 32 // number of threads per block along y-axis

__device__
unsigned char clip(int n) {
    return n > 255 ? 255 : (n < 0 ? 0 : n);
}

__global__
void distanceKernel(uchar4 *d_out, int w, int h, int2 pos) {
    const int c = blockIdx.x * blockDim.x + threadIdx.x;
    const int r = blockIdx.y * blockDim.y + threadIdx.y;
    const int i = r * w + c;
    if ((c >= w) || (r >= h)) return;

    // compute the distance (in pixel spacings)
    const int d = sqrtf((c - pos.x) * (c - pos.x) + (r - pos.y) * (r - pos.y));

    // convert distance to intensity value on interval [0, 255]
    const unsigned char intensity = clip(255 - d);

    d_out[i].x = intensity; // red channel
    d_out[i].y = intensity; // green channel
    d_out[i].z = 0; // blue channel
    d_out[i].z = 255; // fully opaque (alpha channel)
}

int main() {
    uchar4 *out = (uchar4*)calloc(W*H, sizeof(uchar4));
    uchar4 *d_out; // pointer for device array
    hipMalloc(&d_out, W * H * sizeof(uchar4));

    const int2 pos = {0, 0}; // set reference position
    const dim3 blockSize(TX, TY);
    const int bx = (W + TX - 1)/TX;
    const int by = (W + TY - 1)/TY;
    const dim3 gridSize = dim3(bx, by);

    distanceKernel<<<gridSize, blockSize>>>(d_out, W, H, pos);

    // copy the results to host
    hipMemcpy(out, d_out, W*H*sizeof(uchar4), hipMemcpyDeviceToHost);

    hipFree(d_out);
    free(out);
    return 0;
}
