#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>
#include <time.h>
#define TPB 32
#define M 100       // number of times to do the data transfer

__device__
float distance(float x1, float x2) {
    return sqrt((x2-x1)*(x2-x1));
}

__global__
void distanceKernel(float *d_out, float *d_in, float ref) {
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    const float x = d_in[i];
    d_out[i] = distance(x, ref);
}

void distanceArray(float *out, float *in, float ref, int len) {

    // create event variables for timing
    hipEvent_t startMemcpy, stopMemcpy;
    hipEvent_t startKernel, stopKernel;
    hipEventCreate(&startMemcpy);
    hipEventCreate(&stopMemcpy);
    hipEventCreate(&startKernel);
    hipEventCreate(&stopKernel);

    float *d_in = 0;
    float *d_out = 0;
    hipMalloc(&d_in, len*sizeof(float));
    hipMalloc(&d_out, len*sizeof(float));

    // record the event that "starts the clock" on data transfer
    hipEventRecord(startMemcpy);

    // copy input data from host to device M times
    for (int i = 0; i < M; ++i) {
        hipMemcpy(d_in, in, len*sizeof(float), hipMemcpyHostToDevice);
    }

    // record the event that "stops the clock" on data transfer
    hipEventRecord(stopMemcpy);

    hipEventRecord(startKernel);
    distanceKernel<<<len/TPB, TPB>>>(d_out, d_in, ref);
    hipEventRecord(stopKernel);

    hipMemcpy(out, d_out, len*sizeof(float), hipMemcpyDeviceToHost);

    // ensured timed events have stopped
    hipEventSynchronize(stopMemcpy);
    hipEventSynchronize(stopKernel);

    // Convert event records to time and output
    float memcpyTimeInMs = 0;
    hipEventElapsedTime(&memcpyTimeInMs, startMemcpy, stopMemcpy);
    float kernelTimeInMs = 0;
    hipEventElapsedTime(&kernelTimeInMs, startKernel, stopKernel);

    printf("Kernel time (ms): %f\n", kernelTimeInMs);
    printf("Data transfer time (ms): %f\n", memcpyTimeInMs);

    hipFree(d_in);
    hipFree(d_out);
}
