#include "hip/hip_runtime.h"
#include "kernel.h"
#include <hip/hip_runtime_api.h>
#define TPB 32
#define M 100       // number of times to do the data transfer

__device__
float distance(float x1, float x2) {
    return sqrt((x2-x1)*(x2-x1));
}

__global__
void distanceKernel(float *d_out, float *d_in, float ref) {
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    const float x = d_in[i];
    d_out[i] = distance(x, ref);
}

void distanceArray(float *out, float *in, float ref, int len) {

    float *d_in = 0;
    float *d_out = 0;
    checkCudaErrors(hipMalloc(&d_in, len*sizeof(float)));
    checkCudaErrors(hipMalloc(&d_out, len*sizeof(float)));

    checkCudaErrors(hipMemcpy(d_in, in, len*sizeof(float), hipMemcpyHostToDevice));

    distanceKernel<<<len/TPB, TPB>>>(d_out, d_in, ref);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(out, d_out, len*sizeof(float), hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_in));
    checkCudaErrors(hipFree(d_out));
}
